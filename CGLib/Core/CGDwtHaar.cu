#include "hip/hip_runtime.h"
//author 2015 Wang Xinbo

#include "CGDwtHaar.h"

#ifndef COMPILE_WITHOUT_CUDA

using namespace CG;
using namespace CG::Core;

texture<float, 2, hipReadModeElementType> texHaar;

//__global__ void CGConvRow(unsigned char *d_Result, int dataW, int dataH)
__global__ void CGHaarRow(float *d_Result, int dataW, int dataH, int midSize, int stride)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int globalPos = y * stride + x;
	float norm = INV_SQRT_2;

	if(x<dataW && y<dataH)
	{
		if(x < midSize)
		{
			d_Result[globalPos] = (tex2D(texHaar, 2 * x + 1, y) + tex2D(texHaar, 2 * x, y)) * norm;
		}
		else
		{
			d_Result[globalPos] = (tex2D(texHaar, 2 * (x - midSize) + 1, y) - tex2D(texHaar, 2 * (x - midSize), y)) * norm;
		}
	}
	////����
	//if(0 == globalPos%1000)
	//{
	//	printf("*%d %.2f", globalPos, d_Result[globalPos]);
	//}

}

//Y����
__global__ void CGHaarColumn ( float *d_Result, int dataW, int dataH, int midSize, int stride)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int globalPos = y * stride + x;
	float norm = INV_SQRT_2;

	if(x<dataW && y<dataH)
	{
		if(y < midSize)
		{
			d_Result[globalPos] = (tex2D(texHaar, x, 2 * y + 1) + tex2D(texHaar, x, 2 * y)) * norm;
		}
		else
		{
			d_Result[globalPos] = (tex2D(texHaar, x, 2 * (y - midSize) + 1) - tex2D(texHaar, x, 2 * (y - midSize))) * norm;
		}
	}
}

__host__ void 
Core::CGDwtHaar_CUDA(CGImage<float> *ImgDst, CGImage<float> *ImgIn, int haar_level)
{
	dim3 gridSize((ImgIn->width + 16 - 1) / 16, (ImgIn->hight + 16 - 1) / 16);
	dim3 blockSize(16, 16);

	//������
	const hipChannelFormatDesc desc = hipCreateChannelDesc<float>();
	texHaar.normalized = false;
	float *haarBufferRow;
	float *haarBufferCol;
	size_t pitch;
	hipMallocPitch((void**)&haarBufferRow, &pitch, ImgIn->width*sizeof(float), ImgIn->hight);
	hipMallocPitch((void**)&haarBufferCol, &pitch, ImgIn->width*sizeof(float), ImgIn->hight);
	hipMemcpy2D(haarBufferRow, pitch, ImgIn->GetData(true), ImgIn->width*sizeof(float), ImgIn->width*sizeof(float), ImgIn->hight, hipMemcpyDeviceToDevice);

	int stride = pitch/sizeof(float);

	for(int i = 0; i < haar_level; i++)
	{
		hipBindTexture2D(0, texHaar, haarBufferRow, desc, ImgIn->width, ImgIn->hight, pitch);

		CGHaarRow<<<gridSize, blockSize>>>(haarBufferCol, ImgIn->width, ImgIn->hight, ImgIn->width/2, stride);

		hipBindTexture2D(0, texHaar, haarBufferCol, desc, ImgIn->width, ImgIn->hight, pitch);

		CGHaarColumn<<<gridSize, blockSize>>>(haarBufferRow, ImgIn->width, ImgIn->hight, ImgIn->hight/2, stride);
	}

	hipMemcpy2D(ImgDst->GetData(true), ImgIn->width*sizeof(float), haarBufferRow, pitch, ImgIn->width*sizeof(float), ImgIn->hight, hipMemcpyDeviceToDevice);


	hipFree(haarBufferRow);
	hipFree(haarBufferCol);
	hipUnbindTexture(texHaar);
}

#endif