#include "hip/hip_runtime.h"
//author 2015 Wang Xinbo

#include "CGDwtHaar.h"

#ifndef COMPILE_WITHOUT_CUDA

using namespace CG;
using namespace CG::Core;

texture<float, 2, hipReadModeElementType> texHaar;

//__global__ void CGConvRow(unsigned char *d_Result, int dataW, int dataH)
__global__ void CGHaarRow(float *d_Result, int dataW, int dataH, int midSize)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int globalPos = y * dataW + x;
	float norm = INV_SQRT_2;

	if(x<dataW - 1 && y<dataH)
	{
		if(x < midSize)
		{
			d_Result[globalPos] = (tex2D(texHaar, 2 * x + 1, y) + tex2D(texHaar, 2 * x, y)) * norm;
		}
		else
		{
			d_Result[globalPos] = (tex2D(texHaar, 2 * (x - midSize) + 1, y) - tex2D(texHaar, 2 * (x - midSize), y)) * norm;
		}
	}
	////����
	//if(0 == globalPos%1000)
	//{
	//	printf("*%d %.2f", globalPos, d_Result[globalPos]);
	//}

}

//Y����
__global__ void CGHaarColumn ( float *d_Result, int dataW, int dataH, int midSize)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int globalPos = y * dataW + x;
	float norm = INV_SQRT_2;

	if(x<dataW && y<dataH)
	{
		if(y < midSize)
		{
			d_Result[globalPos] = (tex2D(texHaar, x, 2 * y + 1) + tex2D(texHaar, x, 2 * y)) * norm;
		}
		else
		{
			d_Result[globalPos] = (tex2D(texHaar, x, 2 * (y - midSize) + 1) - tex2D(texHaar, x, 2 * (y - midSize))) * norm;
		}
	}
}

__host__ void 
Core::CGDwtHaar_CUDA(CGImage<float> *ImgDst, CGImage<float> *ImgIn)
{
	dim3 gridSize((ImgIn->width + 16 - 1) / 16, (ImgIn->hight + 16 - 1) / 16);
	dim3 blockSize(16, 16);

	//������
	const hipChannelFormatDesc desc = hipCreateChannelDesc<float>();
	texHaar.normalized = false;
	float *haarBuffer;
	size_t pitch;
	hipMallocPitch((void**)&haarBuffer, &pitch, ImgIn->width*sizeof(float), ImgIn->hight);
	hipMemcpy2D(haarBuffer, pitch, ImgIn->GetData(true), ImgIn->width*sizeof(float), ImgIn->width*sizeof(float), ImgIn->hight, hipMemcpyDeviceToDevice);

	//int midSize = ImgIn->dataSize/2;
	int level = HAAR_LEVEL;

	for(int i = 0; i < level; i++)
	{
		hipBindTexture2D(0, texHaar, haarBuffer, desc, ImgIn->width, ImgIn->hight, pitch);

		CGHaarRow<<<gridSize, blockSize>>>(haarBuffer, ImgIn->width, ImgIn->hight, ImgIn->width/2);

		hipBindTexture2D(0, texHaar, haarBuffer, desc, ImgIn->width, ImgIn->hight, pitch);

		if(i < level - 1)
		{
			CGHaarColumn<<<gridSize, blockSize>>>(haarBuffer, ImgIn->width, ImgIn->hight, ImgIn->hight/2);
		}
		else
		{
			CGHaarColumn<<<gridSize, blockSize>>>(ImgDst->GetData(true), ImgIn->width, ImgIn->hight, ImgIn->hight/2);
		}
	}

	hipFree(haarBuffer);
	hipUnbindTexture(texHaar);
}

#endif