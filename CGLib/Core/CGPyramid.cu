#include "hip/hip_runtime.h"
//author 2015 Wang Xinbo

#include "CGPyramid.h"

#ifndef COMPILE_WITHOUT_CUDA

using namespace CG;
using namespace CG::Core;

texture<float, 2, hipReadModeElementType> tex;

__global__ void fastResize(float *result, int width, int hight, int dstWidth, int dstHight, float scale)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int globalPos = y * width + x;

	float u = x*scale;
	float v = y*scale;

	if(x<width && y<hight && x<dstWidth && y<dstHight)
		result[globalPos] = tex2D(tex, u, v);

	//if(x<20 && y<10)
	//{
	//	printf("*%.2f %d ", result[globalPos], x);
	//}
}

__host__ void 
Core::CGPyramid_CUDA(CGImage<float> *ImgDst, CGImage<float> *ImgIn, float scale)
{
	int newWidth = (int)(ImgIn->width/scale);
	int newHight = (int)(ImgIn->hight/scale);
	dim3 gridSize((newWidth + 16 - 1) / 16, (newHight + 16 - 1) / 16);
	dim3 blockSize(16, 16);
	//������
	const hipChannelFormatDesc desc = hipCreateChannelDesc<float>();
	tex.filterMode = hipFilterModeLinear;
	tex.normalized = false;
	float *inTex;
	size_t pitch;
	hipMallocPitch((void**)&inTex, &pitch, ImgIn->width*sizeof(float), ImgIn->hight);
	hipMemcpy2D(inTex, pitch, ImgIn->GetData(true), ImgIn->width*sizeof(float), ImgIn->width*sizeof(float), ImgIn->hight, hipMemcpyDeviceToDevice);
	hipBindTexture2D(0, tex, inTex, desc, ImgIn->width, ImgIn->hight, pitch);

	ImgDst->Clear();
	//ImgDst->ChangeSize(newWidth, newHight);
	fastResize<<<gridSize, blockSize>>>(ImgDst->GetData(true), ImgDst->width, ImgDst->hight, newWidth, newHight, scale);

	hipUnbindTexture(tex);
	hipFree(inTex);
}

#endif